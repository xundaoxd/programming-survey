#include "hip/hip_runtime.h"
#include <iostream>

#include "common.h"

__global__ void inc(float *num, int n) {
  auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  auto idx = tid;
  while (idx < n) {
    num[idx] += 1;
    idx += gridDim.x * blockDim.x;
  }
}

template <std::size_t N> void do_zerocopy() {
  hipDeviceProp_t prop;
  nv_assert(hipGetDeviceProperties(&prop, 0));
  if (!prop.canMapHostMemory) {
    std::cout << "can not map host memory" << std::endl;
    return;
  }
  nv_assert(hipSetDeviceFlags(hipDeviceMapHost));

  float *a_h, *a_map;
  nv_assert(hipHostAlloc(&a_h, sizeof(float) * N, hipHostMallocMapped));
  MAKE_DEFER(hipHostFree(a_h));
  nv_assert(hipHostGetDevicePointer(&a_map, a_h, 0));
  std::fill_n(a_h, N, 0);

  hipEvent_t st, ed;
  nv_assert(hipEventCreate(&st));
  MAKE_DEFER(hipEventDestroy(st));
  nv_assert(hipEventCreate(&ed));
  MAKE_DEFER(hipEventDestroy(ed));

  inc<<<1, 1>>>(a_map, N);
  nv_assert(hipDeviceSynchronize());
  inc<<<1, 1>>>(a_map, N);
  nv_assert(hipEventRecord(st, 0));
  for (int i = 0; i < 100; i++) {
    inc<<<1, 1>>>(a_map, N);
  }
  nv_assert(hipEventRecord(ed, 0));
  nv_assert(hipDeviceSynchronize());
  float tm;
  nv_assert(hipEventElapsedTime(&tm, st, ed));
  std::cout << "zerocopy time " << tm << "ms" << std::endl;
}

template <std::size_t N> void do_normal() {
  float a_h[N];
  std::fill_n(a_h, N, 0);
  float *a_map;
  nv_assert(hipMalloc(&a_map, sizeof(a_h)));
  MAKE_DEFER(hipFree(a_map));
  nv_assert(hipMemcpy(a_map, a_h, sizeof(a_h), hipMemcpyHostToDevice));

  hipEvent_t st, ed;
  nv_assert(hipEventCreate(&st));
  MAKE_DEFER(hipEventDestroy(st));
  nv_assert(hipEventCreate(&ed));
  MAKE_DEFER(hipEventDestroy(ed));

  inc<<<1, 1>>>(a_map, N);
  nv_assert(hipDeviceSynchronize());
  inc<<<1, 1>>>(a_map, N);
  nv_assert(hipEventRecord(st, 0));
  for (int i = 0; i < 100; i++) {
    inc<<<1, 1>>>(a_map, N);
  }
  nv_assert(hipEventRecord(ed, 0));
  nv_assert(hipDeviceSynchronize());
  float tm;
  nv_assert(hipEventElapsedTime(&tm, st, ed));
  std::cout << "normal time " << tm << "ms" << std::endl;
}

int main(int argc, char *argv[]) {
  do_zerocopy<4096>();
  do_normal<4096>();
  return 0;
}
