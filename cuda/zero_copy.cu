#include "hip/hip_runtime.h"
#include <iostream>

#include "common.h"

__global__ void inc(float *num) { *num += 1; }

int main(int argc, char *argv[]) {
  hipDeviceProp_t prop;
  nv_assert(hipGetDeviceProperties(&prop, 0));
  if (!prop.canMapHostMemory) {
    std::cout << "can not map host memory" << std::endl;
    return 0;
  }
  nv_assert(hipSetDeviceFlags(hipDeviceMapHost));

  float *a_h, *a_map;
  nv_assert(hipHostAlloc(&a_h, sizeof(float), hipHostMallocMapped));
  *a_h = 1;
  nv_assert(hipHostGetDevicePointer(&a_map, a_h, 0));

  inc<<<1, 1>>>(a_map);
  nv_assert(hipDeviceSynchronize());

  std::cout << *a_h << std::endl;
  return 0;
}
