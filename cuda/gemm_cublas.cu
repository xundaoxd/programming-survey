#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <algorithm>
#include <cassert>
#include <vector>

#include "hipblas.h"
#include "hip/hip_runtime.h"

#define CUDA_ASSERT(expr)                                                      \
  do {                                                                         \
    auto flag = (expr);                                                        \
    assert(flag == hipSuccess);                                               \
  } while (0)

#define CUBLAS_ASSERT(expr)                                                    \
  do {                                                                         \
    auto flag = (expr);                                                        \
    assert(flag == HIPBLAS_STATUS_SUCCESS);                                     \
  } while (0)

template <typename T, int M, int N, int K> void do_test() {
  class Generator {
    T init_;

  public:
    Generator(T val) : init_(val) {}
    T operator()() { return init_++; }
  };

  std::vector<T> C(M * N);
  std::vector<T> A(M * K);
  std::generate(A.begin(), A.end(), Generator(1));
  std::vector<T> B(K * N);
  std::generate(B.begin(), B.end(), Generator(1));

  T *dev_c;
  T *dev_a;
  T *dev_b;
  CUDA_ASSERT(hipMalloc(&dev_c, sizeof(T) * M * N));

  CUDA_ASSERT(hipMalloc(&dev_a, sizeof(T) * M * K));
  CUDA_ASSERT(hipMalloc(&dev_b, sizeof(T) * K * N));

  hipblasHandle_t handle;
  CUBLAS_ASSERT(hipblasCreate(&handle));

  CUDA_ASSERT(
      hipMemcpy(dev_a, A.data(), sizeof(T) * M * K, hipMemcpyHostToDevice));
  CUDA_ASSERT(
      hipMemcpy(dev_b, B.data(), sizeof(T) * K * N, hipMemcpyHostToDevice));

  float alpha = 1, beta = 0;
  for (int i = 0; i < 1024; i++) {
    CUBLAS_ASSERT(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
                              dev_b, N, dev_a, K, &beta, dev_c, N));
  }

  CUDA_ASSERT(
      hipMemcpy(C.data(), dev_c, sizeof(T) * M * N, hipMemcpyDeviceToHost));
  CUDA_ASSERT(hipDeviceSynchronize());

  CUBLAS_ASSERT(hipblasDestroy(handle));
  CUDA_ASSERT(hipFree(dev_a));
  CUDA_ASSERT(hipFree(dev_b));
  CUDA_ASSERT(hipFree(dev_c));
}

int main(void) {
  do_test<float, 1024, 1024, 1024>();
  return 0;
}
