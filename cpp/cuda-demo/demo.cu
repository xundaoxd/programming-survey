#include "hip/hip_runtime.h"
#include <vector>

#include "common.h"

__global__ void inc(int *nums, int l) {
  for (auto idx = threadIdx.x + blockIdx.x * blockDim.x; idx < l;
       idx += gridDim.x * blockDim.x) {
    nums[idx]++;
  }
}

int main(int argc, char *argv[]) {
  (void)argc;
  (void)argv;
  std::vector<int> nums(1024);
  int *d_nums;
  nv_assert(hipMalloc(&d_nums, sizeof(int) * nums.size()));
  nv_assert(hipMemcpy(d_nums, nums.data(), sizeof(int) * nums.size(),
                       hipMemcpyHostToDevice));
  inc<<<256, 256>>>(d_nums, nums.size());
  nv_assert(hipMemcpy(nums.data(), d_nums, sizeof(int) * nums.size(),
                       hipMemcpyDeviceToHost));
  nv_assert(hipDeviceSynchronize());
  nv_assert(hipFree(d_nums));
  return 0;
}
